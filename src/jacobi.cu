#include "hip/hip_runtime.h"
#include "jacobi.h"

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "config.h"

using namespace std;
using namespace Eigen;

__global__ void jacobi_on_device(const int *outptr, const int *inptr, const double *valptr,
                                 const size_t rows, const size_t cols, const size_t nnz,
                                 const double *b, const double *x_curr, double *x_next) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if ( i < rows ) {
    double diag = 0.0;
    double temp = b[i];
    for (long int cnt = outptr[i]; cnt < outptr[i+1]; ++cnt) {
      if ( inptr[cnt] == i )
        diag = valptr[cnt];
      else
        temp -= x_curr[inptr[cnt]]*valptr[cnt];
    }
    x_next[i] = temp/diag;
  }
}

static void select_gpu(int *gpu_num, int *num_devs) {
  // gpu_num: (I/O): I: Default choice,
  //                 O: best device, changed only if more than one device
  // num_devs: (O)   Number of found devices.
  int best = *gpu_num;
  hipGetDeviceCount(num_devs);

  if ( *num_devs > 1 ) {
    int dev_num;
    int max_cores = 0;
    for (dev_num = 0; dev_num < *num_devs; dev_num++) {
      hipDeviceProp_t dev_properties;
      hipGetDeviceProperties(&dev_properties, dev_num);
      if (max_cores < dev_properties.multiProcessorCount) {
        max_cores = dev_properties.multiProcessorCount;
        best = dev_num;
      }
    }
    *gpu_num = best;
  }
}

static void test_device(int devID) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, devID);
  if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
    printf("There is no device supporting CUDA.\n");
    hipDeviceReset();
  }
  else
    printf("Using GPU device number %d.\n", devID);
}

cuda_jacobi_solver::cuda_jacobi_solver(const SparseMatrix<double, RowMajor> &A)
  : rows_(A.rows()), cols_(A.cols()), nnz_(A.nonZeros()) {
  // allocate memory
  hipMalloc((void **)&outptr_d_, (rows_+1)*sizeof(int));
  hipMalloc((void **)&inptr_d_,  nnz_*sizeof(int));
  hipMalloc((void **)&valptr_d_, nnz_*sizeof(double));
  hipMalloc((void **)&b_d_,      rows_*sizeof(double));
  hipMalloc((void **)&x_curr_d_, cols_*sizeof(double));
  hipMalloc((void **)&x_next_d_, cols_*sizeof(double));
  // copy the system matrix
  ASSERT(hipMemcpy(outptr_d_, A.outerIndexPtr(), (rows_+1)*sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
  ASSERT(hipMemcpy(inptr_d_,  A.innerIndexPtr(), nnz_*sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
  ASSERT(hipMemcpy(valptr_d_, A.valuePtr(),      nnz_*sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
}

cuda_jacobi_solver::~cuda_jacobi_solver() {
  hipFree(outptr_d_);
  hipFree(inptr_d_);
  hipFree(valptr_d_);
  hipFree(b_d_);
  hipFree(x_curr_d_);
  hipFree(x_next_d_);
}

int cuda_jacobi_solver::apply(const VectorXd &b, VectorXd &x) {
  ASSERT(hipMemcpy(b_d_,      b.data(), rows_*sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
  ASSERT(hipMemcpy(x_curr_d_, x.data(), cols_*sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
  x /= 0.0;
  ASSERT(hipMemcpy(x_next_d_, x.data(), cols_*sizeof(double), hipMemcpyHostToDevice) == hipSuccess);

  int blockSize = 1024;
  int nBlocks = 2;
  jacobi_on_device<<<nBlocks, blockSize>>>(outptr_d_, inptr_d_, valptr_d_,
                                           rows_, cols_, nnz_,
                                           b_d_, x_curr_d_, x_next_d_);
  hipDeviceSynchronize();
  hipMemcpy(x.data(), x_next_d_, cols_*sizeof(double), hipMemcpyDeviceToHost);
  return 0;
}
